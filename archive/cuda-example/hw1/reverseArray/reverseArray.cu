#include "hip/hip_runtime.h"
/*
 * cuda_reverse_aray.cpp
 *
 *  Created on: Apr 3, 2009
 *      Author: qzg
 */

// includes, system
#include <stdio.h>
#include <assert.h>
#include "hip/hip_runtime.h"

// Simple utility function to check for CUDA runtime errors
void checkCUDAError(const char* msg);

// Part 2 of 2: implement the kernel
__global__ void reverseArrayBlock(int *d_out, int *d_in)
{
	extern __shared__ int d_in_share[];
	int index=blockDim.x*blockIdx.x+threadIdx.x;
	//////copy data to share mem from global mem
	d_in_share[threadIdx.x]=d_in[index];

	////sync is must
	__syncthreads();
	//////////////////////////////////////////
	////copy data to global mem from share mem
	int out_index=(gridDim.x-blockIdx.x-1)*blockDim.x+threadIdx.x;
	d_out[out_index]=d_in_share[blockDim.x-threadIdx.x-1];

}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main( int argc, char** argv)
{
    // pointer for host memory and size
    int *h_a;
    int dimA = 256 * 1024; // 256K elements (1MB total)

    // pointer for device memory
    int *d_b, *d_a;

    // define grid and block size
    int numThreadsPerBlock = 256;

    // Compute number of blocks needed based on array size and desired block size
    int numBlocks = dimA / numThreadsPerBlock;

    // Part 1 of 2: Compute the number of bytes of shared memory needed
    // This is used in the kernel invocation below
    int sharedMemSize = numThreadsPerBlock*sizeof(int);

    // allocate host and device memory
    size_t memSize = numBlocks * numThreadsPerBlock * sizeof(int);
    h_a = (int *) malloc(memSize);
    hipMalloc( (void **) &d_a, memSize );
    hipMalloc( (void **) &d_b, memSize );

    // Initialize input array on host
    for (int i = 0; i < dimA; ++i)
    {
        h_a[i] = i;
    }

    // Copy host array to device array
    hipMemcpy( d_a, h_a, memSize, hipMemcpyHostToDevice );

    // launch kernel
    dim3 dimGrid(numBlocks);
    dim3 dimBlock(numThreadsPerBlock);
    reverseArrayBlock<<< dimGrid, dimBlock, sharedMemSize >>>( d_b, d_a );

    // block until the device has completed
    hipDeviceSynchronize();

    // check if kernel execution generated an error
    // Check for any CUDA errors
    checkCUDAError("kernel invocation");

    // device to host copy
    hipMemcpy( h_a, d_b, memSize, hipMemcpyDeviceToHost );

    // Check for any CUDA errors
    checkCUDAError("memcpy");

    // verify the data returned to the host is correct
    for (int i = 0; i < dimA; i++)
    {
        assert(h_a[i] == dimA - 1 - i );
    }

    // free device memory
    hipFree(d_a);
    hipFree(d_b);

    // free host memory
    free(h_a);

    // If the program makes it this far, then the results are correct and
    // there are no run-time errors.  Good work!
    printf("Correct!\n");

    return 0;
}

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}
