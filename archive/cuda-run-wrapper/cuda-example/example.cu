#include <stdio.h>
#include <unistd.h>
#include "hip/hip_runtime.h"

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err){
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
        exit(-1);
    }                         
}

__global__ void myFirstKernel(int *d_a  )
{
        int i= blockIdx.x;
        int j=threadIdx.x;
        d_a[i * blockDim.x + j] += 1000 * i + j;
}

int main(int argc, char** argv) 
{
    int *h_a;
    int *d_a;
    int numBlocks = 256;
    int numThreadsPerBlock = 256;
    size_t memSize = numBlocks * numThreadsPerBlock * sizeof(int)* 64;
    int device;
    
    hipGetDevice(&device);
    printf("enter cuda program\n");
    printf("device: %d\n", device);
    checkCUDAError("hipSetDevice");
    
    h_a = (int *)malloc(memSize);
    hipMalloc((void**)&d_a, memSize);
    checkCUDAError("hipMalloc");
    hipMemcpy(d_a, h_a, memSize, hipMemcpyHostToDevice);
    checkCUDAError("hipMemcpy");
    
    sleep(1);
    myFirstKernel<<<numBlocks,  numThreadsPerBlock>>>(d_a);
    checkCUDAError("kernel execution");

    sleep(1);
    hipMemcpy(h_a, d_a, memSize, hipMemcpyDeviceToHost); 
    checkCUDAError("hipMemcpy");
    
    hipFree(d_a);
    free(h_a);
    return 0;
}
